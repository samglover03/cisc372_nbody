#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

__global__ void compute_kernel(double* hPos, double* hVel, double* mass, vector3* accels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i == j){
        FILL_VECTOR(accels[i * NUMENTITIES + j], 0, 0, 0);
        return;
    }
    vector3 distance;
	for (int k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
	double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
	double magnitude=sqrt(magnitude_sq);
	double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
	FILL_VECTOR(accels[i * NUMENTITIES + j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
}

__global__ void sum(vector3 *accels, vector3 *accel_sum, vector3 *dPos, vector3 *dVel) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < NUMENTITIES) {
        FILL_VECTOR(accel_sum[i], 0, 0, 0);
        for (int j = 0; j < NUMENTITIES; j++) {
            for (int k = 0; k < 3; k++) {
                accel_sum[i][k] += accels[(i * NUMENTITIES) + j][k];
            }
        }
        // Compute the new velocity based on the acceleration and time interval
        // Compute the new position based on the velocity and time interval
        for (int k = 0; k < 3; k++) {
            dVel[i][k] += accel_sum[i][k] * INTERVAL;
            dPos[i][k] += dVel[i][k] * INTERVAL; 
        }
    }
}

void compute() {
    double *dmass;
    vector3 *dhPos, *dhVel, *dacc, *dsum;

	int block = ceilf(NUMENTITIES / 16.0f);
	int thread = ceilf(NUMENTITIES / (float) block);

	dim3 gridDim(block, block, 1);
	dim3 blockDim(thread, thread, 1);

	hipMalloc((void**) &dmass, sizeof(double) * NUMENTITIES);
	hipMalloc((void**) &dhPos, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**) &dhVel, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**) &dacc, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**) &dsum, sizeof(vector3) * NUMENTITIES);
	
    hipMemcpy(dmass, mass, sizeof(double)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(dhPos, hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(dhVel, hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
	
	compute_kernel<<<gridDim, blockDim>>>(dhPos, dhVel, dmass, dacc);
	hipDeviceSynchronize();

	sum<<<gridDim.x, blockDim.x>>>(dacc, dsum, dhPos, dhVel);
	hipDeviceSynchronize();

	hipMemcpy(hPos, dhPos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hVel, dhVel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);

	hipFree(dhPos);
	hipFree(dhVel);
	hipFree(dmass);
	hipFree(dacc);
}